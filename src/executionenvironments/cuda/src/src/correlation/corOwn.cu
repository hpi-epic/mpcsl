#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "corOwn.cuh"
#include "corHelper.cuh"
#include "../util/indepUtil.h"

#define NUMTHREADS 32
#define PERTHREAD 8

void gpuPMCC(const double * h_mat, int n,
    int dim, double * h_cors) {
  if (VERBOSE)
    printf("Cor started with N=%i, dim=%i\n", n, dim);
  size_t
    dbytes = sizeof(double);
  double
    *d_mat, *d_means, *d_stddevs,
    *d_cors;
  dim3
    block(NUMTHREADS), grid(n, n), gridX(n);

  hipMalloc(reinterpret_cast<void **>(&d_means), n * dbytes);
  hipMalloc(reinterpret_cast<void **>(&d_stddevs), n * dbytes);
  hipMalloc(reinterpret_cast<void **>(&d_cors), n*n*dbytes);

  hipMalloc(reinterpret_cast<void **>(&d_mat), n*dim*dbytes);
  hipMemcpy(d_mat, h_mat, n*dim*dbytes, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  gpuMeans<<<gridX, block>>>(d_mat, n, dim, d_means);
  hipDeviceSynchronize();
  if (VERBOSE)
    printf("Means successful \n");

  gpuSD<<<gridX, block>>>(d_mat, n, dim, d_means, d_stddevs);
  hipDeviceSynchronize();
  if (VERBOSE)
    printf("SD successful \n");

  gpuPMCC<<<grid, block>>>(d_mat, n, dim, d_means, d_stddevs, d_cors);
  hipMemcpy(h_cors, d_cors, n*n*dbytes,
    hipMemcpyDeviceToHost);
  if (VERBOSE)
    printf("PMCC successful \n");

  // Free allocated space
  hipFree(d_means);
  hipFree(d_stddevs);
  hipFree(d_cors);
  hipFree(d_mat);
}


void gpuPMCCShared(const double * h_mat, int n,
    int dim, double * h_cors) {
  size_t
    dbytes = sizeof(double);
  double
    *d_mat, *d_means, *d_stddevs,
    *d_cors;
  size_t gridY = ((n%PERTHREAD == 0) ? n/PERTHREAD : (n/PERTHREAD) + 1);
  dim3
    block(NUMTHREADS), grid(n, gridY), gridX(n);
  hipMalloc(reinterpret_cast<void **>(&d_means), n * dbytes);
  hipMalloc(reinterpret_cast<void **>(&d_stddevs), n * dbytes);
  hipMalloc(reinterpret_cast<void **>(&d_cors), n*n*dbytes);

  hipMalloc(reinterpret_cast<void **>(&d_mat), n*dim*dbytes);
  hipMemcpy(d_mat, h_mat, n*dim*dbytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  gpuMeans<<<gridX, block>>>(d_mat, n, dim, d_means);
  hipDeviceSynchronize();

  gpuSD<<<gridX, block>>>(d_mat, n, dim, d_means, d_stddevs);
  hipDeviceSynchronize();

  gpuPMCCShared<<<grid, block>>>(d_mat, n, dim, d_means, d_stddevs, d_cors);
  hipMemcpy(h_cors, d_cors, n*n*dbytes,
    hipMemcpyDeviceToHost);

  // Free allocated space
  hipFree(d_means);
  hipFree(d_stddevs);
  hipFree(d_cors);
  hipFree(d_mat);
}
