#include "hip/hip_runtime.h"
#include "testL0.cuh"
#include "indepTests.cuh"
#include <chrono>

TestResult indTestL0(State h_state) {
    double* d_adj, *d_cor, *d_pMax;
    int* d_sepSets;
    hipMalloc(reinterpret_cast<void **> (&d_adj),
               sizeof(double) * h_state.p * h_state.p);
    hipMalloc(reinterpret_cast<void **> (&d_cor),
               sizeof(double) * h_state.p * h_state.p);
    hipMalloc(reinterpret_cast<void **> (&d_pMax),
               sizeof(double) * h_state.p * h_state.p);
    hipMalloc(reinterpret_cast<void **> (&d_sepSets),
               sizeof(int) * h_state.p * h_state.p);
    hipMemcpy(d_adj, h_state.adj, sizeof(double) * h_state.p * h_state.p,
               hipMemcpyHostToDevice);
    hipMemcpy(d_cor, h_state.cor, sizeof(double) * h_state.p * h_state.p,
               hipMemcpyHostToDevice);
    hipMemcpy(d_sepSets, h_state.sepSets,
               sizeof(int) * h_state.p * h_state.p,
               hipMemcpyHostToDevice);
    State d_state = { d_pMax, d_adj, d_cor, d_sepSets, h_state.p,
                      h_state.observations, h_state.alpha,
                      h_state.maxCondSize };
    int numthreads = min(d_state.p, 32);
    dim3 block(numthreads), grid(d_state.p);
    auto start = std::chrono::system_clock::now();

    testL0<<<grid, block>>>(d_state);
    hipDeviceSynchronize();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>
        (std::chrono::system_clock::now() - start).count();
    hipMemcpy(h_state.pMax, d_state.pMax,
               sizeof(double) * h_state.p * h_state.p, hipMemcpyDeviceToHost);
    hipMemcpy(h_state.adj, d_state.adj,
               sizeof(double) * h_state.p * h_state.p, hipMemcpyDeviceToHost);
    hipMemcpy(h_state.sepSets, d_state.sepSets,
               sizeof(int) * h_state.p * h_state.p,
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_sepSets);
    hipFree(d_adj);
    hipFree(d_cor);
    hipFree(d_pMax);
    return { static_cast<unsigned long>(duration),
             (h_state.p * (h_state.p - 1)) / 2 };
}
