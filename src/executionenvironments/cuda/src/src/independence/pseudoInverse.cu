#include "hip/hip_runtime.h"
#include "pseudoInverse.cuh"
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>

void calculateMatrixPseudoInverse(double *tmpMat, double *inverse, int kSize) {
    double *d_U, *d_VT, *d_S, *d_S_inv;

    hipMalloc(reinterpret_cast<void **> (&d_U),
               sizeof(double) * kSize * kSize);
    hipMalloc(reinterpret_cast<void **> (&d_VT),
               sizeof(double) * kSize * kSize);
    hipMalloc(reinterpret_cast<void **> (&d_S),
               sizeof(double) * kSize);
    hipMalloc(reinterpret_cast<void **> (&d_S_inv),
               sizeof(double) * kSize * kSize);

    svd(tmpMat, kSize, d_U, d_VT, d_S);

    dim3 block(kSize), gridX(kSize);
    matrixInverse<<<gridX, block>>>(d_S, d_S_inv, kSize);
    hipDeviceSynchronize();

    matrixMatrixMultiplication(d_VT, d_S_inv, d_U, kSize, inverse);

    hipFree(d_U);
    hipFree(d_VT);
    hipFree(d_S);
    hipFree(d_S_inv);
}

void svd(double *tmpMat, int kSize, double *d_U, double *d_VT, double *d_S) {
    hipsolverHandle_t cusolverH;
    int lwork = 0;
    double *d_work, *d_rwork, *d_A;
    int *devInfo = NULL;
    char jobu = 'A';
    char jobvt = 'A';


    hipMalloc(reinterpret_cast<void **> (&d_A),
               sizeof(double) * kSize * kSize);
    hipMalloc(reinterpret_cast<void **> (&devInfo), sizeof(int));
    hipMemcpy(d_A, tmpMat, sizeof(double) * kSize * kSize,
               hipMemcpyHostToDevice);

    hipsolverDnCreate(&cusolverH);
    hipsolverDnDgesvd_bufferSize(cusolverH, kSize, kSize, &lwork);

    hipMalloc(reinterpret_cast<void **> (&d_work), sizeof(double)*lwork);
    hipMalloc(reinterpret_cast<void **> (&d_rwork), sizeof(double)*lwork);
    hipsolverDnDgesvd(cusolverH, jobu, jobvt, kSize, kSize,
        d_A, kSize, d_S, d_U, kSize, d_VT,
        kSize, d_work, lwork, d_rwork, devInfo);

    hipFree(d_work);
    hipFree(d_rwork);
    hipFree(d_A);
    hipFree(devInfo);
}

void matrixMatrixMultiplication(double *d_VT, double * d_S_inv,
                                double *d_U, int kSize, double *inverse) {
    const double factor = 1.0;
    const double beta = 0.0;
    double *tmpRes, *result;

    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);

    hipMalloc(reinterpret_cast<void **> (&tmpRes),
               sizeof(double) * kSize * kSize);
    hipMalloc(reinterpret_cast<void **> (&result),
               sizeof(double) * kSize * kSize);

    // cublas matmul transpose(VT) * s_inv = tmpRes
    hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, kSize, kSize, kSize, &factor,
        d_VT, kSize, d_S_inv, kSize, &beta, tmpRes, kSize);

    // cublas matmul tmpRes * transpose(U) = tmpMat
    hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, kSize, kSize, kSize, &factor,
        tmpRes, kSize, d_U, kSize, &beta, result, kSize);
    hipDeviceSynchronize();

    hipMemcpy(inverse, result, sizeof(double)*kSize * kSize,
               hipMemcpyDeviceToHost);
    hipFree(tmpRes);
    hipFree(result);
}

__global__ void matrixInverse(double *S, double *S_inv, int kSize) {
    int bx = blockIdx.x, tx = threadIdx.x;

    if (bx == tx) {
        S_inv[bx * kSize + tx] = 1.f / S[bx];
    } else {
        S_inv[bx * kSize + tx] = 0.f;
    }
}
